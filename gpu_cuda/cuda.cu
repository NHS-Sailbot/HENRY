#include "hip/hip_runtime.h"
#include "cuda.hpp"

static constexpr unsigned int 
    PIXEL_SIZE = 3 * sizeof(unsigned char), 
    WIDTH = 1280, HEIGHT = 720, 
    PIXEL_COUNT = WIDTH * HEIGHT, 
    DATA_SIZE = PIXEL_COUNT * PIXEL_SIZE;

static constexpr dim3 BLOCK_SIZE(10, 10), THREAD_SIZE(128, 72);

__global__ void pixel_kernel(unsigned char *const out, unsigned char *const pixel) {
    const unsigned int x = blockIdx.x * THREAD_SIZE.x + threadIdx.x;
    const unsigned int y = blockIdx.y * THREAD_SIZE.y + threadIdx.y;
    const unsigned int index = (x + y * WIDTH) * PIXEL_SIZE;
    // if (index < DATA_SIZE - 2) {
        out[index + 0] = 255;
        out[index + 1] = 0;
        out[index + 2] = 255;
    // }
}

namespace cuda {
    static unsigned char *data1 = nullptr, *data2 = nullptr;

    void init() {
        hipMalloc(&data1, DATA_SIZE);
        hipMalloc(&data2, DATA_SIZE);
    }
    void perform_kernel(unsigned char *const p) {
        hipMemcpy(data1, p, DATA_SIZE, hipMemcpyHostToDevice);
        hipMemcpy(data2, p, DATA_SIZE, hipMemcpyHostToDevice);
        pixel_kernel<<<BLOCK_SIZE, THREAD_SIZE>>>(data1, data2);
        hipMemcpy(p, data1, DATA_SIZE, hipMemcpyDeviceToHost);
        // for (unsigned int i = 0; i < DATA_SIZE; ++i) p[i] = 100;
    }
    void deinit() {
        hipFree(data1);
        hipFree(data2);
    }
}
